#include "hip/hip_runtime.h"
﻿
#include "etc.h"

#include <stdio.h>
//#include <exception>
#include <stdexcept>

#include "hip/hip_runtime.h"
#include ""

//#include <hipcub/hipcub.hpp>

#include "sh_rot.cuh"

// Gaussian splatting's SH definition is different from
// wiki https://en.wikipedia.org/wiki/Table_of_spherical_harmonics
// Enable this flag to do transformation
#define NEG_SH_SIGN

// DEBUG
//#include "../cnpy/cnpy.h"


#define MY_SURF_BOUNDARY_MODE hipBoundaryModeTrap

__forceinline__ __device__ float sigmoid(float x)
{
	return 1.0f / (1.0f + expf(-x));
}

__forceinline__ __device__ float length4(float* x) {
	return sqrt(x[0] * x[0] + x[1] * x[1] + x[2] * x[2] + x[3] * x[3]);
}

__forceinline__ __device__ int padding_shared_memory(int x, int n_pad = 1) {
	return x + (x / 32 * n_pad);
}

__device__ void quaternion_raw_multiply(
	float* c,
	const float* a,
	const float* b
) {
	c[0] = a[0] * b[0] - a[1] * b[1] - a[2] * b[2] - a[3] * b[3];
	c[1] = a[0] * b[1] + a[1] * b[0] + a[2] * b[3] - a[3] * b[2];
	c[2] = a[0] * b[2] - a[1] * b[3] + a[2] * b[0] + a[3] * b[1];
	c[3] = a[0] * b[3] + a[1] * b[2] - a[2] * b[1] + a[3] * b[0];
}

__device__ void quaternion_to_matrix(float* mat, const float* quat) {
	float r = quat[0];
	float i = quat[1];
	float j = quat[2];
	float k = quat[3];
	float two_s = 2.f / (r * r + i * i + j * j + k * k);
	mat[0] = 1.f - two_s * (j * j + k * k);
	mat[1] = two_s * (i * j - k * r);
	mat[2] = two_s * (i * k + j * r);
	mat[3] = two_s * (i * j + k * r);
	mat[4] = 1.f - two_s * (i * i + k * k);
	mat[5] = two_s * (j * k - i * r);
	mat[6] = two_s * (i * k - j * r);
	mat[7] = two_s * (j * k + i * r);
	mat[8] = 1.f - two_s * (i * i + j * j);
}

__device__ void matrix_to_quaternion(float * outq, const float* mat) {	
	float q_abs[4];
	q_abs[0] = 1 + mat[0] + mat[4] + mat[8];
	q_abs[1] = 1 + mat[0] - mat[4] - mat[8];
	q_abs[2] = 1 - mat[0] + mat[4] - mat[8];
	q_abs[3] = 1 - mat[0] - mat[4] + mat[8];
#pragma unroll
	for (int i = 0; i < 4; i++) {
		if (q_abs[i] > 0) q_abs[i] = sqrt(q_abs[i]);
		else q_abs[i] = 0;
	}
	float div;
	int argmax_id = 0;
	float max_v = q_abs[0];
	if (q_abs[1] > max_v) {	max_v = q_abs[1]; argmax_id = 1;}
	if (q_abs[2] > max_v) { max_v = q_abs[2]; argmax_id = 2;}
	if (q_abs[3] > max_v) { max_v = q_abs[3]; argmax_id = 3;}
	if (argmax_id == 0) {		
		outq[0] = q_abs[0] * q_abs[0];
		outq[1] = mat[7] - mat[5];
		outq[2] = mat[2] - mat[6];
		outq[3] = mat[3] - mat[1];
	}
	else if (argmax_id == 1) {
		outq[0] = mat[7] - mat[5];
		outq[1] = q_abs[1] * q_abs[1];
		outq[2] = mat[3] + mat[1];
		outq[3] = mat[2] + mat[6];
	}
	else if (argmax_id == 2) {
		outq[0] = mat[2] - mat[6];
		outq[1] = mat[3] + mat[1];
		outq[2] = q_abs[2] * q_abs[2];
		outq[3] = mat[5] + mat[7];
	}else{
		outq[0] = mat[3] - mat[1];
		outq[1] = mat[6] + mat[2];
		outq[2] = mat[7] + mat[5];
		outq[3] = q_abs[3] * q_abs[3];
	}
	div = q_abs[argmax_id]; // div 0 should never happen ?
	//if (div < 0.1f) div = 0.1f;
	div *= 2;
	outq[0] /= div;
	outq[1] /= div;
	outq[2] /= div;
	outq[3] /= div;
}

__device__ void inv_of_mat3x3(const float* src, float* tar) {

	float det = src[0] * (src[4] * src[8] - src[7] * src[5]) -
		src[1] * (src[3] * src[8] - src[5] * src[6]) +
		src[2] * (src[3] * src[7] - src[4] * src[6]);
	float invdet = 1.f / det;
	tar[0] = (src[4] * src[8] - src[7] * src[5]) * invdet;
	tar[1] = (src[2] * src[7] - src[1] * src[8]) * invdet;
	tar[2] = (src[1] * src[5] - src[2] * src[4]) * invdet;
	tar[3] = (src[5] * src[6] - src[3] * src[8]) * invdet;
	tar[4] = (src[0] * src[8] - src[2] * src[6]) * invdet;
	tar[5] = (src[3] * src[2] - src[0] * src[5]) * invdet;
	tar[6] = (src[3] * src[7] - src[6] * src[4]) * invdet;
	tar[7] = (src[6] * src[1] - src[0] * src[7]) * invdet;
	tar[8] = (src[0] * src[4] - src[3] * src[1]) * invdet;
}

__device__ void transpose_mat3x3(const float* src, float* tar) {
	float tmp;
	tar[0] = src[0]; tar[1] = src[3]; tar[2] = src[6];
	tar[3] = src[1]; tar[4] = src[4]; tar[5] = src[7];
	tar[6] = src[2]; tar[7] = src[5]; tar[8] = src[8];
}

__device__ void normalize_transform(const float* src, float* tar, int N = 5) {

	float* mat = tar;
	float mat_T[9];
	float mat_Tinv[9];
	for (int j = 0; j < 9; j++) {
		mat[j] = src[j];
	}

	for (int i = 0; i < N; i++) {
		//float mat_next[9];
		transpose_mat3x3(mat, mat_T);
		inv_of_mat3x3(mat_T, mat_Tinv);
		for (int j = 0; j < 9; j++) {
			mat[j] = 0.5f * (mat[j] + mat_Tinv[j]);
		}
	}
}

__device__ void colormap_jet(float v, float* color){
	if (v < 0.0) v = 0.0;
	if (v > 1.0) v = 1.0;

	color[0] = 0.f;
	color[1] = 0.f;
	color[2] = 0.f;
	// G
	if (v < 0.125f) {
		color[1] = 0.f;
	}
	else if (v < 0.375f) {
		color[1] = (v - 0.125f) / (0.375f - 0.125f);
	}
	else if (v < 0.64f) {
		color[1] = 1.f;
	}
	else if (v < 0.91f) {
		color[1] = 1.f - (v - 0.64f) / (0.91f - 0.64f);
	}
	// R
	if (v < 0.35f) {
		color[0] = 0.f;
	}
	else if (v < 0.66f) {
		color[0] = (v - 0.35f) / (0.66f - 0.35f);
	}
	else if (v < 0.888f) {
		color[0] = 1.f;
	}
	else {
		color[0] = 0.5f + 4.464285714285714f * (1.f - v);
	}
	// B
	if (v < 0.112f) {
		color[2] = 0.5f + 4.464285714285714f * v;
	}
	else if (v < 0.34f) {
		color[2] = 1.f;
	}
	else if (v < 0.65f) {
		color[2] = 1.f - (v - 0.34f) / (0.65f - 0.34f);
	}
}

//////////////////////////////

template<int THREADS>
__global__ void CopyRotKernel(
	int64_t n_elements,
	const float* __restrict__ src,
	float* __restrict__ tar,
	bool use_activation
) {
	__shared__ float sdata[THREADS * 4 + (THREADS * 4) / 32];
#pragma unroll
	for (int i = 0; i < 4; i++) {
		int loc_offset = i * THREADS + threadIdx.x;
		int64_t i_elem = blockIdx.x * (THREADS * 4) + loc_offset;
		if (i_elem < n_elements * 4) {
			sdata[padding_shared_memory(loc_offset)] = src[i_elem];
		}
	}
	__syncthreads();
	int64_t i_elem = blockIdx.x * THREADS + threadIdx.x;
	float loc_rot[4];
	if (i_elem < n_elements) {
		loc_rot[0] = sdata[padding_shared_memory(threadIdx.x * 4 + 0)];
		loc_rot[1] = sdata[padding_shared_memory(threadIdx.x * 4 + 1)];
		loc_rot[2] = sdata[padding_shared_memory(threadIdx.x * 4 + 2)];
		loc_rot[3] = sdata[padding_shared_memory(threadIdx.x * 4 + 3)];
		if (use_activation) {
			float length = length4(loc_rot);
			loc_rot[0] /= length;
			loc_rot[1] /= length;
			loc_rot[2] /= length;
			loc_rot[3] /= length;
		}
	}
	__syncthreads();
	if (i_elem < n_elements) {
		sdata[padding_shared_memory(threadIdx.x * 4 + 0)] = loc_rot[0];
		sdata[padding_shared_memory(threadIdx.x * 4 + 1)] = loc_rot[1];
		sdata[padding_shared_memory(threadIdx.x * 4 + 2)] = loc_rot[2];
		sdata[padding_shared_memory(threadIdx.x * 4 + 3)] = loc_rot[3];
	}
	__syncthreads();
#pragma unroll
	for (int i = 0; i < 4; i++) {
		int loc_offset = i * THREADS + threadIdx.x;
		int64_t i_elem = blockIdx.x * (THREADS * 4) + loc_offset;
		if (i_elem < n_elements * 4) {
			tar[i_elem] = sdata[padding_shared_memory(loc_offset)];
		}
	}
}

template<int P>
__global__ void CopyScaleKernel(
	int64_t n_elements, 
	const float * __restrict__ src,
	float * __restrict__ tar,
	bool use_activation
) {
	for (int i_loop = 0; i_loop < P; i_loop++) {
		int64_t i_elem = ((int64_t)blockIdx.x * P + i_loop) * blockDim.x + (int64_t)threadIdx.x;
		if (i_elem < n_elements) {
			float v = src[i_elem];
			if (use_activation) {
				v = __expf(v);
			}
			tar[i_elem] = v;
		}
	}
}

template<int P>
__global__ void CopyOpacityKernel(
	int64_t n_elements,
	const float* __restrict__ src,
	float* __restrict__ tar,
	bool use_activation
) {
	for (int i_loop = 0; i_loop < P; i_loop++) {
		int64_t i_elem = ((int64_t)blockIdx.x * P + i_loop) * blockDim.x + (int64_t)threadIdx.x;
		if (i_elem < n_elements) {
			float v = src[i_elem];
			if (use_activation) {
				v = sigmoid(v);
			}
			tar[i_elem] = v;
		}
	}
}


__global__ void TransferPosKernel(
	int64_t n_elements,
	const float * __restrict__ mat4x3,
	const float * __restrict__ src,
	float * __restrict__ tar
) {
	__shared__ float smat[12];
	if (threadIdx.x < 12) {
		smat[threadIdx.x] = mat4x3[threadIdx.x];
	}
	__syncthreads();
	float loc_mat[12];
	for (int i = 0; i < 12; i++) {
		loc_mat[i] = smat[i];
	}
	int64_t i_elem = blockIdx.x * blockDim.x + threadIdx.x;
	if (i_elem < n_elements) {
		float src_pos[3];
		float tar_pos[3];
		src_pos[0] = src[3 * i_elem + 0];
		src_pos[1] = src[3 * i_elem + 1];
		src_pos[2] = src[3 * i_elem + 2];
		tar_pos[0] = loc_mat[0] * src_pos[0] + loc_mat[1] * src_pos[1] + loc_mat[2] * src_pos[2] + loc_mat[3];
		tar_pos[1] = loc_mat[4] * src_pos[0] + loc_mat[5] * src_pos[1] + loc_mat[6] * src_pos[2] + loc_mat[7];
		tar_pos[2] = loc_mat[8] * src_pos[0] + loc_mat[9] * src_pos[1] + loc_mat[10] * src_pos[2] + loc_mat[11];
		tar[3 * i_elem + 0] = tar_pos[0];
		tar[3 * i_elem + 1] = tar_pos[1];
		tar[3 * i_elem + 2] = tar_pos[2];		
	}
}

__global__ void TransferRotActivationKernel(
	int64_t n_elements,
	const float* __restrict__ qrot4,
	const float* __restrict__ src,
	float* __restrict__ tar
) {
	__shared__ float srot[4];
	if (threadIdx.x < 4) {
		srot[threadIdx.x] = qrot4[threadIdx.x];
	}
	__syncthreads();
	float loc_rot[4];
	for (int i = 0; i < 4; i++) {
		loc_rot[i] = srot[i];
	}
	int64_t i_elem = blockIdx.x * blockDim.x + threadIdx.x;
	if (i_elem < n_elements) {
		float v[4];
		v[0] = src[i_elem * 4 + 0];
		v[1] = src[i_elem * 4 + 1];
		v[2] = src[i_elem * 4 + 2];
		v[3] = src[i_elem * 4 + 3];
		float length = length4(v);
		v[0] = v[0] / length;
		v[1] = v[1] / length;
		v[2] = v[2] / length;
		v[3] = v[3] / length;
		// do rot
		float tmpv[4];
		tmpv[0] = v[0]; tmpv[1] = v[1]; tmpv[2] = v[2]; tmpv[3] = v[3];
		quaternion_raw_multiply(v, loc_rot, tmpv);
		tar[i_elem * 4 + 0] = v[0];
		tar[i_elem * 4 + 1] = v[1];
		tar[i_elem * 4 + 2] = v[2];
		tar[i_elem * 4 + 3] = v[3];
	}
}

// threads = 32 * 3
// shared memory for sh = 32 * 3 * 16
// shared memory for matrix = 32 * 9
__global__ void __launch_bounds__(32 * 3) TransferSHKernel(
	int64_t n_elements,
	const float* __restrict__ mat3x3,
	const float* __restrict__ src,
	float* __restrict__ tar
) {
	__shared__ float s_shs[32 * 3 * 16];
	__shared__ float s_mat[9];
#pragma unroll
	for (int i = 0; i < 16; i++) {
		int loc_offset = i * 96 + threadIdx.x;
		int64_t i_elem = blockIdx.x * (96 * 16) + loc_offset;
		if (i_elem < n_elements * 48) {
			s_shs[loc_offset] = src[i_elem];
		}
	}
	if (threadIdx.x < 9) {
		s_mat[threadIdx.x] = mat3x3[threadIdx.x];
	}
	__syncthreads();
	int64_t thread_idx = blockIdx.x * 96 + threadIdx.x;
	int64_t i_elem = thread_idx / 3;
	int li_elem = threadIdx.x / 3;
	int channel = threadIdx.x % 3;
	float loc_sh_out[16];
	if (i_elem < n_elements) {
		float loc_sh[16];
		float loc_mat[9];
		// copy sh to local
#pragma unroll
		for (int i = 0; i < 16; i++) {
			loc_sh[i] = s_shs[li_elem * 48 + i * 3 + channel];
		}
#ifdef NEG_SH_SIGN
#pragma unroll
		for (int i = 0; i < 16; i++) {
			if (i % 2) loc_sh[i] = -loc_sh[i];
		}
#endif
		// copy transform matrix to local
		for (int i = 0; i < 9; i++) {
			loc_mat[i] = s_mat[i];
		}
		float sh1[3][3];
		float sh2[5][5];
		float sh3[7][7];
		Construct_SH_Rotation_Matrix(loc_mat, sh1, sh2, sh3);
		loc_sh_out[0] = loc_sh[0];

		loc_sh_out[1] = sh1[0][0] * loc_sh[1] + sh1[0][1] * loc_sh[2] + sh1[0][2] * loc_sh[3];
		loc_sh_out[2] = sh1[1][0] * loc_sh[1] + sh1[1][1] * loc_sh[2] + sh1[1][2] * loc_sh[3];
		loc_sh_out[3] = sh1[2][0] * loc_sh[1] + sh1[2][1] * loc_sh[2] + sh1[2][2] * loc_sh[3];
#pragma unroll
		for (int i = 0; i < 5; i++) {
			loc_sh_out[4 + i] = 0;
#pragma unroll
			for (int j = 0; j < 5; j++) {
				loc_sh_out[4 + i] += sh2[i][j] * loc_sh[4 + j];
			}
		}
#pragma unroll
		for (int i = 0; i < 7; i++) {
			loc_sh_out[9 + i] = 0;
#pragma unroll
			for (int j = 0; j < 7; j++) {
				loc_sh_out[9 + i] += sh3[i][j] * loc_sh[9 + j];
			}
		}

#ifdef NEG_SH_SIGN
#pragma unroll
		for (int i = 0; i < 16; i++) {
			if (i % 2) loc_sh_out[i] = -loc_sh_out[i];
		}
#endif
	}
	//// write back, first to shared memory, then to global memory
	__syncthreads();
	if (i_elem < n_elements) {
#pragma unroll
		for (int i = 0; i < 16; i++) {
			s_shs[li_elem * 48 + i * 3 + channel] = loc_sh_out[i];
		}
	}
	__syncthreads();
#pragma unroll
	for (int i = 0; i < 16; i++) {
		int loc_offset = i * 96 + threadIdx.x;
		int64_t i_elem = blockIdx.x * (96 * 16) + loc_offset;
		if (i_elem < n_elements * 48) {
			tar[i_elem] = s_shs[loc_offset];
		}
	}
	
}

// threads = 32 * 3
// shared memory for sh = 32 * 3 * 16
// shared memory for matrix = 32 * 9
__global__ void __launch_bounds__(32 * 3) TransferSHKernelQuat(
	int64_t n_elements,
	const float* __restrict__ quatT, // 4xP
	const float* __restrict__ src, // Px16x3
	float * __restrict__ tar // Px16x3
) {
	__shared__ float s_shs[32 * 3 * 16];
#pragma unroll
	for (int i = 0; i < 16; i++) {
		int loc_offset = i * 96 + threadIdx.x;
		int64_t i_elem = blockIdx.x * (96 * 16) + loc_offset;
		if (i_elem < n_elements * 48) {
			s_shs[loc_offset] = src[i_elem];
		}
	}
	__syncthreads();
	int64_t thread_idx = blockIdx.x * 96 + threadIdx.x;
	int64_t i_elem = thread_idx / 3;
	int li_elem = threadIdx.x / 3;
	int channel = threadIdx.x % 3;
	float loc_sh_out[16];
	if (i_elem < n_elements) {
		float loc_sh[16];
		float loc_quat[4];
		float loc_mat[9];
		// copy sh to local
#pragma unroll
		for (int i = 0; i < 16; i++) {
			loc_sh[i] = s_shs[li_elem * 48 + i * 3 + channel];
		}
#ifdef NEG_SH_SIGN
#pragma unroll
		for (int i = 0; i < 16; i++) {
			if (i % 2) loc_sh[i] = -loc_sh[i];
		}
#endif
		// copy transform matrix to local
		//for (int i = 0; i < 9; i++) {
		//	loc_mat[i] = s_mat[i];
		//}
		loc_quat[0] = quatT[n_elements * 0 + i_elem];
		loc_quat[1] = quatT[n_elements * 1 + i_elem];
		loc_quat[2] = quatT[n_elements * 2 + i_elem];
		loc_quat[3] = quatT[n_elements * 3 + i_elem];
		quaternion_to_matrix(loc_mat, loc_quat);

		float sh1[3][3];
		float sh2[5][5];
		float sh3[7][7];
		Construct_SH_Rotation_Matrix(loc_mat, sh1, sh2, sh3);
		loc_sh_out[0] = loc_sh[0];

		loc_sh_out[1] = sh1[0][0] * loc_sh[1] + sh1[0][1] * loc_sh[2] + sh1[0][2] * loc_sh[3];
		loc_sh_out[2] = sh1[1][0] * loc_sh[1] + sh1[1][1] * loc_sh[2] + sh1[1][2] * loc_sh[3];
		loc_sh_out[3] = sh1[2][0] * loc_sh[1] + sh1[2][1] * loc_sh[2] + sh1[2][2] * loc_sh[3];
#pragma unroll
		for (int i = 0; i < 5; i++) {
			loc_sh_out[4 + i] = 0;
#pragma unroll
			for (int j = 0; j < 5; j++) {
				loc_sh_out[4 + i] += sh2[i][j] * loc_sh[4 + j];
			}
		}
#pragma unroll
		for (int i = 0; i < 7; i++) {
			loc_sh_out[9 + i] = 0;
#pragma unroll
			for (int j = 0; j < 7; j++) {
				loc_sh_out[9 + i] += sh3[i][j] * loc_sh[9 + j];
			}
		}

#ifdef NEG_SH_SIGN
#pragma unroll
		for (int i = 0; i < 16; i++) {
			if (i % 2) loc_sh_out[i] = -loc_sh_out[i];
		}
#endif
	}
	//// write back, first to shared memory, then to global memory
	__syncthreads();
	if (i_elem < n_elements) {
#pragma unroll
		for (int i = 0; i < 16; i++) {
			s_shs[li_elem * 48 + i * 3 + channel] = loc_sh_out[i];
		}
	}
	__syncthreads();
#pragma unroll
	for (int i = 0; i < 16; i++) {
		int loc_offset = i * 96 + threadIdx.x;
		int64_t i_elem = blockIdx.x * (96 * 16) + loc_offset;
		if (i_elem < n_elements * 48) {
			tar[i_elem] = s_shs[loc_offset];
		}
	}
}

// threads = 32 * 3
// shared memory for sh = 32 * 3 * 16
// shared memory for matrix = 32 * 9
__global__ void __launch_bounds__(32 * 3) TransferSHKernelQuat_Inplace(
	int64_t n_elements,
	const float* __restrict__ quatT, // 4xP
	float* __restrict__ in_out // Px16x3
) {
	__shared__ float s_shs[32 * 3 * 16];
#pragma unroll
	for (int i = 0; i < 16; i++) {
		int loc_offset = i * 96 + threadIdx.x;
		int64_t i_elem = blockIdx.x * (96 * 16) + loc_offset;
		if (i_elem < n_elements * 48) {
			s_shs[loc_offset] = in_out[i_elem];
		}
	}
	__syncthreads();
	int64_t thread_idx = blockIdx.x * 96 + threadIdx.x;
	int64_t i_elem = thread_idx / 3;
	int li_elem = threadIdx.x / 3;
	int channel = threadIdx.x % 3;
	float loc_sh_out[16];
	if (i_elem < n_elements) {
		float loc_sh[16];
		float loc_quat[4];
		float loc_mat[9];
		// copy sh to local
#pragma unroll
		for (int i = 0; i < 16; i++) {
			loc_sh[i] = s_shs[li_elem * 48 + i * 3 + channel];
		}
#ifdef NEG_SH_SIGN
#pragma unroll
		for (int i = 0; i < 16; i++) {
			if (i % 2) loc_sh[i] = -loc_sh[i];
		}
#endif
		// copy transform matrix to local
		//for (int i = 0; i < 9; i++) {
		//	loc_mat[i] = s_mat[i];
		//}
		loc_quat[0] = quatT[n_elements * 0 + i_elem];
		loc_quat[1] = quatT[n_elements * 1 + i_elem];
		loc_quat[2] = quatT[n_elements * 2 + i_elem];
		loc_quat[3] = quatT[n_elements * 3 + i_elem];
		quaternion_to_matrix(loc_mat, loc_quat);

		float sh1[3][3];
		float sh2[5][5];
		float sh3[7][7];
		Construct_SH_Rotation_Matrix(loc_mat, sh1, sh2, sh3);
		loc_sh_out[0] = loc_sh[0];

		loc_sh_out[1] = sh1[0][0] * loc_sh[1] + sh1[0][1] * loc_sh[2] + sh1[0][2] * loc_sh[3];
		loc_sh_out[2] = sh1[1][0] * loc_sh[1] + sh1[1][1] * loc_sh[2] + sh1[1][2] * loc_sh[3];
		loc_sh_out[3] = sh1[2][0] * loc_sh[1] + sh1[2][1] * loc_sh[2] + sh1[2][2] * loc_sh[3];
#pragma unroll
		for (int i = 0; i < 5; i++) {
			loc_sh_out[4 + i] = 0;
#pragma unroll
			for (int j = 0; j < 5; j++) {
				loc_sh_out[4 + i] += sh2[i][j] * loc_sh[4 + j];
			}
		}
#pragma unroll
		for (int i = 0; i < 7; i++) {
			loc_sh_out[9 + i] = 0;
#pragma unroll
			for (int j = 0; j < 7; j++) {
				loc_sh_out[9 + i] += sh3[i][j] * loc_sh[9 + j];
			}
		}

#ifdef NEG_SH_SIGN
#pragma unroll
		for (int i = 0; i < 16; i++) {
			if (i % 2) loc_sh_out[i] = -loc_sh_out[i];
		}
#endif
	}
	//// write back, first to shared memory, then to global memory
	__syncthreads();
	if (i_elem < n_elements) {
#pragma unroll
		for (int i = 0; i < 16; i++) {
			s_shs[li_elem * 48 + i * 3 + channel] = loc_sh_out[i];
		}
	}
	__syncthreads();
#pragma unroll
	for (int i = 0; i < 16; i++) {
		int loc_offset = i * 96 + threadIdx.x;
		int64_t i_elem = blockIdx.x * (96 * 16) + loc_offset;
		if (i_elem < n_elements * 48) {
			in_out[i_elem] = s_shs[loc_offset];
		}
	}
}

// FUFace edition
template<int SN>
__global__ void CompositeBasisPosKernel(
	int64_t n_elements, int p_offset, int p_n, int p_total,
	const float * __restrict__ params, // expr(51)
	const float * __restrict__ pos_mean, // {Px3}
	const float * __restrict__ pos_basis, // {Px3}xK expr
	float * __restrict__ pos_tar // {Px3}
) {
	__shared__ float tile[SN];
	if (threadIdx.x < p_n) {
		tile[threadIdx.x] = params[p_offset + threadIdx.x];
	}
	__syncthreads();
	int64_t i_elem = (int64_t)blockIdx.x * blockDim.x + (int64_t)threadIdx.x;
	if (i_elem < n_elements) {
		float v[3];
		v[0] = pos_mean[i_elem * 3 + 0];
		v[1] = pos_mean[i_elem * 3 + 1];
		v[2] = pos_mean[i_elem * 3 + 2];
		for (int ii = 0; ii < p_n; ii++) { // expr basis
			float k = tile[ii];
			int real_i = ii + p_offset;
			v[0] += pos_basis[(i_elem * 3 + 0) * p_total + real_i] * k;
			v[1] += pos_basis[(i_elem * 3 + 1) * p_total + real_i] * k;
			v[2] += pos_basis[(i_elem * 3 + 2) * p_total + real_i] * k;
		}
		pos_tar[i_elem * 3 + 0] = v[0];
		pos_tar[i_elem * 3 + 1] = v[1];
		pos_tar[i_elem * 3 + 2] = v[2];
	}
}

// FLAME edition
template<int SN>
__global__ void CompositeBasisPosKernel(
	int64_t n_elements, int p_offset, int p_n, int p_total,
	int p_offset2, int p_n2,
	const float* __restrict__ params, // expr(100) + eyelid(2) + flatten_node(36)
	const float* __restrict__ pos_mean, // {Px3}
	const float* __restrict__ pos_basis, // {Px3}xK expr
	const float* __restrict__ pos_basis2, // 36xPx3 pos
	float* __restrict__ pos_tar // {Px3}
) {
	__shared__ float tile[SN];
	if (threadIdx.x < p_n + p_n2) {
		const float* l_ptr;
		if (threadIdx.x < p_n) {
			l_ptr = &params[p_offset + threadIdx.x];
		}
		else {
			l_ptr = &params[p_offset2 + (threadIdx.x - p_n)];
		}
		tile[threadIdx.x] = *l_ptr;
	}
	__syncthreads();
	int64_t i_elem = (int64_t)blockIdx.x * blockDim.x + (int64_t)threadIdx.x;
	if (i_elem < n_elements) {
		float v[3];
		v[0] = pos_mean[i_elem * 3 + 0];
		v[1] = pos_mean[i_elem * 3 + 1];
		v[2] = pos_mean[i_elem * 3 + 2];
		for (int ii = 0; ii < p_n; ii++) { // expr basis
			float k = tile[ii];
			int real_i = ii + p_offset;
			v[0] += pos_basis[(i_elem * 3 + 0) * p_total + real_i] * k;
			v[1] += pos_basis[(i_elem * 3 + 1) * p_total + real_i] * k;
			v[2] += pos_basis[(i_elem * 3 + 2) * p_total + real_i] * k;
		}
		for (int ii = 0; ii < p_n2; ii++) { // pose basis
			float k = tile[p_n + ii];
			v[0] += pos_basis2[(ii * n_elements + i_elem) * 3 + 0] * k;
			v[1] += pos_basis2[(ii * n_elements + i_elem) * 3 + 1] * k;
			v[2] += pos_basis2[(ii * n_elements + i_elem) * 3 + 2] * k;
		}
		pos_tar[i_elem * 3 + 0] = v[0];
		pos_tar[i_elem * 3 + 1] = v[1];
		pos_tar[i_elem * 3 + 2] = v[2];
	}
}


template<int SN>
__global__ void CompositeRotKernel_Inplace(
	int64_t n_elements, 
	const float* __restrict__ params,
	const float* __restrict__ quatT, // 4xP
	float * __restrict__ rot_inout // {Px4}
) {
	int64_t i_elem = (int64_t)blockIdx.x * blockDim.x + (int64_t)threadIdx.x;
	if (i_elem < n_elements) {
		float v[4];
		v[0] = rot_inout[i_elem * 4 + 0];
		v[1] = rot_inout[i_elem * 4 + 1];
		v[2] = rot_inout[i_elem * 4 + 2];
		v[3] = rot_inout[i_elem * 4 + 3];
		float loc_quatT[4];
		loc_quatT[0] = quatT[0 * n_elements + i_elem];
		loc_quatT[1] = quatT[1 * n_elements + i_elem];
		loc_quatT[2] = quatT[2 * n_elements + i_elem];
		loc_quatT[3] = quatT[3 * n_elements + i_elem];
		float tmpv[4];
		tmpv[0] = v[0]; tmpv[1] = v[1]; tmpv[2] = v[2]; tmpv[3] = v[3];
		quaternion_raw_multiply(v, loc_quatT, tmpv);
		rot_inout[i_elem * 4 + 0] = v[0];
		rot_inout[i_elem * 4 + 1] = v[1];
		rot_inout[i_elem * 4 + 2] = v[2];
		rot_inout[i_elem * 4 + 3] = v[3];
	}
}

template<int SN>
__global__ void CompositeBasisRotKernel3(
	int64_t n_elements, int p_offset, int p_n, int p_total, bool enable_rot,
	const float* __restrict__ params,
	const float* __restrict__ rot_mean, // Px4
	const float* __restrict__ rot_basis, // Px4xK 
	const float* __restrict__ ptr_DR, // 4xP
	float* __restrict__ rot_tar // {Px4}
) {
	__shared__ float tile[SN];
	if (threadIdx.x < p_n) {
		tile[threadIdx.x] = params[p_offset + threadIdx.x];
	}
	__syncthreads();
	int64_t i_elem = (int64_t)blockIdx.x * blockDim.x + (int64_t)threadIdx.x;
	if (i_elem < n_elements) {
		float v[4];
		v[0] = rot_mean[i_elem * 4 + 0];
		v[1] = rot_mean[i_elem * 4 + 1];
		v[2] = rot_mean[i_elem * 4 + 2];
		v[3] = rot_mean[i_elem * 4 + 3];

		if (enable_rot) {
			float deform_rot[4];			
			deform_rot[0] = ptr_DR[0 * n_elements + i_elem];
			deform_rot[1] = ptr_DR[1 * n_elements + i_elem];
			deform_rot[2] = ptr_DR[2 * n_elements + i_elem];
			deform_rot[3] = ptr_DR[3 * n_elements + i_elem];

			float tmpv[4];
			tmpv[0] = v[0]; tmpv[1] = v[1]; tmpv[2] = v[2]; tmpv[3] = v[3];
			quaternion_raw_multiply(v, deform_rot, tmpv);
		}
		//////////
		for (int ii = 0; ii < p_n; ii++) {
			float k = tile[ii];
			int real_i = ii + p_offset;
			v[0] += rot_basis[(i_elem * 4 + 0) * p_total + real_i] * k;
			v[1] += rot_basis[(i_elem * 4 + 1) * p_total + real_i] * k;
			v[2] += rot_basis[(i_elem * 4 + 2) * p_total + real_i] * k;
			v[3] += rot_basis[(i_elem * 4 + 3) * p_total + real_i] * k;
		}
		// do normalization
		float length = length4(v);
		rot_tar[i_elem * 4 + 0] = v[0] / length;
		rot_tar[i_elem * 4 + 1] = v[1] / length;
		rot_tar[i_elem * 4 + 2] = v[2] / length;
		rot_tar[i_elem * 4 + 3] = v[3] / length;
	}
}

template<int THREADS>
__global__ void RotPosAndAddEyelidKernel_Inplace(
	int64_t n_elements, int p_offset,
	const float* __restrict__ params, // expr(100) + eyelid(2) + flatten_node(36)
	const float* __restrict__ T, // 12xP
	const float* __restrict__ eyelid, // {P}x3x2
	float* __restrict__ pos_inout
) {
	__shared__ float spos[THREADS * 3];
#pragma unroll
	for (int i = 0; i < 3; i++) {
		int loc_offset = i * THREADS + threadIdx.x;
		int64_t i_elem = blockIdx.x * (THREADS * 3) + loc_offset;
		if (i_elem < n_elements * 3) {
			spos[loc_offset] = pos_inout[i_elem];
		}
	}
	__syncthreads();
	int64_t i_elem = blockIdx.x * THREADS + threadIdx.x;
	float pos[3];
	float pos_out[3];
	if (i_elem < n_elements) {
		pos[0] = spos[threadIdx.x * 3 + 0];
		pos[1] = spos[threadIdx.x * 3 + 1];
		pos[2] = spos[threadIdx.x * 3 + 2];
		//pos_out[0] = pos[0];
		//pos_out[1] = pos[1];
		//pos_out[2] = pos[2];
		float loc_T[12];
		for (int j = 0; j < 12; j++) {
			loc_T[j] = T[j * n_elements + i_elem];
		}
		pos_out[0] = loc_T[0] * pos[0] + loc_T[1] * pos[1] + loc_T[2] * pos[2] + loc_T[3];
		pos_out[1] = loc_T[4] * pos[0] + loc_T[5] * pos[1] + loc_T[6] * pos[2] + loc_T[7];
		pos_out[2] = loc_T[8] * pos[0] + loc_T[9] * pos[1] + loc_T[10] * pos[2] + loc_T[11];
		if (eyelid) {
			float k1 = params[p_offset];
			float k2 = params[p_offset + 1];
			//float k1 = 1.f;
			//float k2 = 0.f;
			pos_out[0] += eyelid[i_elem * 6 + 0] * k1 + eyelid[i_elem * 6 + 1] * k2;
			pos_out[1] += eyelid[i_elem * 6 + 2] * k1 + eyelid[i_elem * 6 + 3] * k2;
			pos_out[2] += eyelid[i_elem * 6 + 4] * k1 + eyelid[i_elem * 6 + 5] * k2;
		}
	}
	__syncthreads();
	if (i_elem < n_elements) {
		spos[threadIdx.x * 3 + 0] = pos_out[0];
		spos[threadIdx.x * 3 + 1] = pos_out[1];
		spos[threadIdx.x * 3 + 2] = pos_out[2];
	}
	__syncthreads();
#pragma unroll
	for (int i = 0; i < 3; i++) {
		int loc_offset = i * THREADS + threadIdx.x;
		int64_t i_elem = blockIdx.x * (THREADS * 3) + loc_offset;
		if (i_elem < n_elements * 3) {
			pos_inout[i_elem] = spos[loc_offset];
		}
	}	
}

template<int SN>
__global__ void CompositeDeformRot(
	int64_t n_elements, int p_offset, int p_n, int p_total, bool do_norm,
	const float* __restrict__ params,
	const float* __restrict__ rot_d, // Px4xK
	float * __restrict__ ptr_DR // 4xP
) {
	__shared__ float tile[SN];
	if (threadIdx.x < p_n) {
		tile[threadIdx.x] = params[p_offset + threadIdx.x];
	}
	__syncthreads();
	int64_t i_elem = (int64_t)blockIdx.x * blockDim.x + (int64_t)threadIdx.x;
	if (i_elem < n_elements) {
		float deform_rot[4] = { 1.f,0.f,0.f,0.f };
		for (int ii = 0; ii < p_n; ii++) {
			float k = tile[ii];
			int real_i = ii + p_offset;
			deform_rot[0] += rot_d[(i_elem * 4 + 0) * p_total + real_i] * k;
			deform_rot[1] += rot_d[(i_elem * 4 + 1) * p_total + real_i] * k;
			deform_rot[2] += rot_d[(i_elem * 4 + 2) * p_total + real_i] * k;
			deform_rot[3] += rot_d[(i_elem * 4 + 3) * p_total + real_i] * k;
		}
		if (do_norm) {
			float length = length4(deform_rot);
			deform_rot[0] = deform_rot[0] / length;
			deform_rot[1] = deform_rot[1] / length;
			deform_rot[2] = deform_rot[2] / length;
			deform_rot[3] = deform_rot[3] / length;
		}
		ptr_DR[0 * n_elements + i_elem] = deform_rot[0];
		ptr_DR[1 * n_elements + i_elem] = deform_rot[1];
		ptr_DR[2 * n_elements + i_elem] = deform_rot[2];
		ptr_DR[3 * n_elements + i_elem] = deform_rot[3];
	}
}

template<int THREADS>
__global__ void CompositeT(
	int64_t n_elements,
	const float* __restrict__ W, // {P}x5
	const float* __restrict__ node_transfer, // 5x4x4
	float* __restrict__ T, // 12xP
	float* __restrict__ quatT // 4xP
) {
	__shared__ float sweight[THREADS * 5];
	__shared__ float snode[5 * 4 * 4];
#pragma unroll
	for (int i = 0; i < 5; i++) {
		int loc_offset = i * THREADS + threadIdx.x;
		int64_t i_elem = blockIdx.x * (THREADS * 5) + loc_offset;
		if (i_elem < n_elements * 5) {
			sweight[loc_offset] = W[i_elem];
		}
	}
	if (threadIdx.x < 5 * 4 * 4) {
		snode[threadIdx.x] = node_transfer[threadIdx.x];
	}
	__syncthreads();
	int64_t i_elem = blockIdx.x * THREADS + threadIdx.x;
	if (i_elem < n_elements) {
		
		float loc_mat[12];
		float loc_rot[4];

		for (int j = 0; j < 12; j++)
			loc_mat[j] = 0.f;
		
		for (int i_node = 0; i_node < 5; i_node++) {
			float w = sweight[threadIdx.x * 5 + i_node];
			for (int j = 0; j < 12; j++) {
				loc_mat[j] += snode[i_node * 16 + j] * w;
			}
		}

		float rot_mat[9];
		float norm_rot_mat[9];
		rot_mat[0] = loc_mat[0]; rot_mat[1] = loc_mat[1]; rot_mat[2] = loc_mat[2];
		rot_mat[3] = loc_mat[4]; rot_mat[4] = loc_mat[5]; rot_mat[5] = loc_mat[6];
		rot_mat[6] = loc_mat[8]; rot_mat[7] = loc_mat[9]; rot_mat[8] = loc_mat[10];
		normalize_transform(rot_mat, norm_rot_mat);
		matrix_to_quaternion(loc_rot, norm_rot_mat);
		// write back
		for (int j = 0; j < 12; j++) {
			T[j * n_elements + i_elem] = loc_mat[j];
		}
		for (int j = 0; j < 4; j++) {
			quatT[j * n_elements + i_elem] = loc_rot[j];
		}
	}
}



////////////////////////////////////////////

// threads = 32 * 3
// shared memory for sh = 32 * 3 * 16
// shared memory for matrix = 32 * 9
__global__ void __launch_bounds__(32*3) CompositeBasisRotSHKernel(
	int64_t n_elements, //int p_offset, int p_n, int p_total,
	const float* __restrict__ faceNR,
	const float* __restrict__ shs_in, // {P}xLx3
	float* __restrict__ shs_out // {P}xLx3
) {
	__shared__ float s_shs[32 * 3 * 16];
	__shared__ float s_mat[32 * 9];
#pragma unroll
	for (int i = 0; i < 16; i++) {
		int loc_offset = i * 96 + threadIdx.x;
		int64_t i_elem = blockIdx.x * (96 * 16) + loc_offset;
		if (i_elem < n_elements * 48) {
			s_shs[loc_offset] = shs_in[i_elem];
		}
	}
#pragma unroll
	for (int i = 0; i < 3; i++) {
		int loc_offset = i * 96 + threadIdx.x;
		int64_t i_elem = blockIdx.x * (96 * 3) + loc_offset;
		if (i_elem < n_elements * 9) {
			s_mat[loc_offset] = faceNR[i_elem];
		}
	}
	__syncthreads();
	int64_t thread_idx = blockIdx.x * 96 + threadIdx.x;
	int64_t i_elem = thread_idx / 3;
	int li_elem = threadIdx.x / 3;
	int channel = threadIdx.x % 3;
	float loc_sh_out[16];
	if (i_elem < n_elements) {		
		float loc_sh[16];
		float loc_mat[9];		
#pragma unroll
		for (int i = 0; i < 16; i++) {
			loc_sh[i] = s_shs[li_elem * 48 + i * 3 + channel];
		}
#ifdef NEG_SH_SIGN
#pragma unroll
		for (int i = 0; i < 16; i++) {
			if (i % 2) loc_sh[i] = -loc_sh[i];
		}
#endif
//#pragma unroll
//		for (int i = 0; i < 9; i++) {
//			loc_mat[i] = s_mat[li_elem * 9 + i];
//		}
// 
		// Eigen Matrix in col major !!
		loc_mat[0] = s_mat[li_elem * 9 + 0]; loc_mat[1] = s_mat[li_elem * 9 + 3]; loc_mat[2] = s_mat[li_elem * 9 + 6];
		loc_mat[3] = s_mat[li_elem * 9 + 1]; loc_mat[4] = s_mat[li_elem * 9 + 4]; loc_mat[5] = s_mat[li_elem * 9 + 7];
		loc_mat[6] = s_mat[li_elem * 9 + 2]; loc_mat[7] = s_mat[li_elem * 9 + 5]; loc_mat[8] = s_mat[li_elem * 9 + 8];

		float sh1[3][3];
		float sh2[5][5];
		float sh3[7][7];
		Construct_SH_Rotation_Matrix(loc_mat, sh1, sh2, sh3);

		/*for (int i = 0; i < 16; i++) {
			loc_sh_out[i] = loc_sh[i];
		}*/
		
		loc_sh_out[0] = loc_sh[0];

		loc_sh_out[1] = sh1[0][0] * loc_sh[1] + sh1[0][1] * loc_sh[2] + sh1[0][2] * loc_sh[3];
		loc_sh_out[2] = sh1[1][0] * loc_sh[1] + sh1[1][1] * loc_sh[2] + sh1[1][2] * loc_sh[3];
		loc_sh_out[3] = sh1[2][0] * loc_sh[1] + sh1[2][1] * loc_sh[2] + sh1[2][2] * loc_sh[3];

#pragma unroll
		for (int i = 0; i < 5; i++) {
			loc_sh_out[4 + i] = 0;
#pragma unroll
			for (int j = 0; j < 5; j++) {
				loc_sh_out[4 + i] += sh2[i][j] * loc_sh[4 + j];
			}
		}

#pragma unroll
		for (int i = 0; i < 7; i++) {
			loc_sh_out[9 + i] = 0;
#pragma unroll
			for (int j = 0; j < 7; j++) {
				loc_sh_out[9 + i] += sh3[i][j] * loc_sh[9 + j];
			}
		}

#ifdef NEG_SH_SIGN
#pragma unroll
		for (int i = 0; i < 16; i++) {
			if (i % 2) loc_sh_out[i] = -loc_sh_out[i];
		}
#endif

	}
	//// write back, first to shared memory, then to global memory
	__syncthreads();
	if (i_elem < n_elements) {
#pragma unroll
		for (int i = 0; i < 16; i++) {
			s_shs[li_elem * 48 + i * 3 + channel] = loc_sh_out[i];
		}
	}
	__syncthreads();
#pragma unroll
	for (int i = 0; i < 16; i++) {
		int loc_offset = i * 96 + threadIdx.x;
		int64_t i_elem = blockIdx.x * (96 * 16) + loc_offset;
		if (i_elem < n_elements * 48) {
			shs_out[i_elem] = s_shs[loc_offset];
		}
	}
}


/////////////////////////////// ACC KERNEL

template<int SN>
__global__ void __launch_bounds__(64) CompositeBasisSHKernelAccN32T64_Inplace(
	int64_t n_elements,
	const float * __restrict__ params,
	float * shs, // {PxLx3}
	const float * __restrict__ shs_basis // {PxLx3}xK
) {
	__shared__ float tile[SN + 32 * SN];
	const int n_load_pass = (SN + 32 * SN + 63) / 64;
	const float* ptr;

	for (int i = 0; i < n_load_pass; i++) {
		int loc_offset = i * 64 + threadIdx.x;
		bool load = true;
		if (loc_offset < SN) {
			ptr = params + loc_offset;
		}
		else {
			int64_t i_elem = (blockIdx.x * 32) * SN + (loc_offset - SN);
			ptr = shs_basis + i_elem;
			if (i_elem >= n_elements * SN || (loc_offset - SN) >= 32 * SN)
				load = false;
		}
		if (load) {
			tile[loc_offset] = *ptr;
		}
	}
	__syncthreads();
	// split 32 elements -> 16(warp0) + 16(warp1) to reduce shared memory bank conflicts
	if (threadIdx.x % 32 < 16) {
		int thread_idx = (threadIdx.x / 32) * 16 + (threadIdx.x % 32);
		int64_t i_elem = blockIdx.x * 32 + thread_idx;
		if (i_elem < n_elements) {
			float v = shs[i_elem];
			for (int ii = 0; ii < SN; ii++) {
				float k = tile[ii];
				float bt = tile[SN + (thread_idx * SN + ii)];
				v += bt * k;
			}
			shs[i_elem] = v;
		}
	}
}


template<int SN>
__global__ void __launch_bounds__(128) CompositeBasisSHKernelAccN128T128_Inplace(
	int64_t n_elements,
	const float* __restrict__ params,
	float* shs, //{PxLx3}
	const float* __restrict__ shs_basis // {PxLx3}xK
) {
	__shared__ float tile[SN + 128 * SN];// TODO: May meet bank conflict, 50% efficiency
	const int n_load_pass = (SN + 128 * SN + 127) / 128;
	const float* ptr;

	for (int i = 0; i < n_load_pass; i++) {
		int loc_offset = i * 128 + threadIdx.x;
		bool load = true;
		if (loc_offset < SN) {
			ptr = params + loc_offset;
		}
		else {
			int64_t i_elem = (blockIdx.x * 128) * SN + (loc_offset - SN);
			ptr = shs_basis + i_elem;
			if (i_elem >= n_elements * SN || (loc_offset - SN) >= 128 * SN)
				load = false;
		}
		if (load) {
			tile[loc_offset] = *ptr;
		}
	}
	__syncthreads();

	int64_t i_elem = blockIdx.x * 128 + threadIdx.x;
	if (i_elem < n_elements) {
		float v = shs[i_elem];
		for (int ii = 0; ii < SN; ii++) {
			float k = tile[ii];
			float bt = tile[SN + (threadIdx.x * SN + ii)];
			v += bt * k;
		}
		shs[i_elem] = v;
	}
}

template<int SN>
__global__ void __launch_bounds__(64) CompositeBasisSHKernelAccN64T64_Inplace(
	int64_t n_elements,
	const float * __restrict__ params,
	float* shs, //{PxLx3}
	const float * __restrict__ shs_basis // {PxLx3}xK
) {
	__shared__ float tile[SN + 64 * SN];// TODO: May meet bank conflict, 50% efficiency
	const int n_load_pass = (SN + 64 * SN + 63) / 64;
	const float* ptr;

	for (int i = 0; i < n_load_pass; i++) {
		int loc_offset = i * 64 + threadIdx.x;
		bool load = true;
		if (loc_offset < SN) {
			ptr = params + loc_offset;
		}
		else {
			int64_t i_elem = (blockIdx.x * 64) * SN + (loc_offset - SN);
			ptr = shs_basis + i_elem;
			if (i_elem >= n_elements * SN || (loc_offset - SN) >= 64 * SN)
				load = false;
		}
		if (load) {
			tile[loc_offset] = *ptr;
		}
	}
	__syncthreads();
		
	int64_t i_elem = blockIdx.x * 64 + threadIdx.x;
	if (i_elem < n_elements) {
		float v = shs[i_elem];
		for (int ii = 0; ii < SN; ii++) {
			float k = tile[ii];
			float bt = tile[SN + (threadIdx.x * SN + ii)];
			v += bt * k;
		}
		shs[i_elem] = v;
	}	
}

template<int SN>
__global__ void __launch_bounds__(64) CompositeBasisSHKernelAccN64T64(
	int64_t n_elements,
	const float* __restrict__ params,
	const float* __restrict__ shs_mean, // {PxLx3}
	const float* __restrict__ shs_basis, // {PxLx3}xK
	float* __restrict__ shs_tar  // {PxLx3}
) {
	__shared__ float tile[SN + 64 * SN];// TODO: May meet bank conflict, 50% efficiency
	const int n_load_pass = (SN + 64 * SN + 63) / 64;
	const float* ptr;
	
	for (int i = 0; i < n_load_pass; i++) {
		int loc_offset = i * 64 + threadIdx.x;
		bool load = true;
		if (loc_offset < SN) {
			ptr = params + loc_offset;
		}
		else {
			int64_t i_elem = (blockIdx.x * 64) * SN + (loc_offset - SN);
			ptr = shs_basis + i_elem;
			if (i_elem >= n_elements * SN || (loc_offset - SN) >= 64 * SN)
				load = false;
		}
		if (load) {
			tile[loc_offset] = *ptr;
		}
	}
	__syncthreads();

	int64_t i_elem = blockIdx.x * 64 + threadIdx.x;
	if (i_elem < n_elements) {
		float v = shs_mean[i_elem];
		for (int ii = 0; ii < SN; ii++) {
			float k = tile[ii];
			float bt = tile[SN + (threadIdx.x * SN + ii)];
			v += bt * k;
		}
		shs_tar[i_elem] = v;
	}
}

template<int SN>
__global__ void __launch_bounds__(256) CompositeBasisSHKernelAccN32T256_Inplace(
	int64_t n_elements, 
	const float* __restrict__ params,
	float* shs, // {PxLx3}
	const float* __restrict__ shs_basis // {PxLx3}xK
) {

	__shared__ float tile[SN + 32 * SN];// TODO: May meet bank conflict, 50% efficiency
	const int n_load_pass = (SN + 32*SN + 255) / 256;
	const float* ptr;
	
	for (int i = 0; i < n_load_pass; i++) {
		int loc_offset = i * 256 + threadIdx.x;
		bool load = true;
		if (loc_offset < SN) {
			ptr = params + loc_offset;
		}
		else {
			int64_t i_elem = (blockIdx.x * 32) * SN + (loc_offset - SN);
			ptr = shs_basis + i_elem;
			if (i_elem >= n_elements * SN || (loc_offset - SN) >= 32 * SN)
				load = false;
		}
		if (load) {
			tile[loc_offset] = *ptr;
		}
	}
	__syncthreads();

	if (threadIdx.x < 32) {
		int64_t i_elem = blockIdx.x * 32 + threadIdx.x;
		if (i_elem < n_elements) {
			float v = shs[i_elem];
			for (int ii = 0; ii < SN; ii++) {
				float k = tile[ii];
				float bt = tile[SN + (threadIdx.x * SN + ii)];
				v += bt * k;
			}
			shs[i_elem] = v;
		}	
	}

}


/////////////////////////////// END ACC KERNEL

template<int P, int SN>
__global__ void CompositeBasisSHKernel_Inplace(
	int64_t n_elements, int p_offset, int p_n, int p_total,
	const float* __restrict__ params,
	float *  shs, // {PxLx3}
	const float* __restrict__ shs_basis // {PxLx3}xK
) {
	__shared__ float tile[SN];
	if (threadIdx.x < p_n) {
		tile[threadIdx.x] = params[p_offset + threadIdx.x];
	}
	__syncthreads();
	for (int i_loop = 0; i_loop < P; i_loop++) {
		int64_t i_elem = ((int64_t)blockIdx.x * P + i_loop) * blockDim.x + (int64_t)threadIdx.x;
		if (i_elem < n_elements) {
			float v = shs[i_elem];
			for (int ii = 0; ii < p_n; ii++) {
				float k = tile[ii];
				int real_i = ii + p_offset;
				v += shs_basis[i_elem * p_total + real_i] * k;
			}
			shs[i_elem] = v;
			//shs_tar[i_elem] = v;
		}
	}
}

template<int P,int SN>
__global__ void CompositeBasisSHKernel(
	int64_t n_elements, int p_offset, int p_n, int p_total,
	const float* __restrict__ params,
	const float* __restrict__ shs_mean, // {PxLx3}
	const float* __restrict__ shs_basis, // {PxLx3}xK
	float * __restrict__ shs_tar  // {PxLx3}
) {
	__shared__ float tile[SN];
	if (threadIdx.x < p_n) {
		tile[threadIdx.x] = params[p_offset + threadIdx.x];
	}
	__syncthreads();
	for (int i_loop = 0; i_loop < P; i_loop++) {
		int64_t i_elem = ((int64_t)blockIdx.x * P + i_loop) * blockDim.x + (int64_t)threadIdx.x;
		if (i_elem < n_elements) {
			float v = shs_mean[i_elem];
			for (int ii = 0; ii < p_n; ii++) {
				float k = tile[ii];
				int real_i = ii + p_offset;
				v += shs_basis[i_elem * p_total + real_i] * k;
			}
			shs_tar[i_elem] = v;
		}
	}
}

template<int P, int SN>
__global__ void CompositeBasisScaleKernel(
	int64_t n_elements, int p_offset, int p_n, int p_total,
	const float* __restrict__ params,
	const float* __restrict__ scale_mean, // {Px3}
	const float* __restrict__ scale_basis, // {Px3}xK
	float* __restrict__ scale_tar  // {Px3}
) {
	__shared__ float tile[SN];
	if (threadIdx.x < p_n) {
		tile[threadIdx.x] = params[p_offset + threadIdx.x];
	}
	__syncthreads();
	for (int i_loop = 0; i_loop < P; i_loop++) {
		int64_t i_elem = ((int64_t)blockIdx.x * P + i_loop) * blockDim.x + (int64_t)threadIdx.x;
		if (i_elem < n_elements) {
			float v = scale_mean[i_elem];
			for (int ii = 0; ii < p_n; ii++) {
				float k = tile[ii];
				int real_i = ii + p_offset;
				v += scale_basis[i_elem * p_total + real_i] * k;
			}
			v = __expf(v);
			scale_tar[i_elem] = v;
			//scale_tar[i_elem] = 0.001f;
		}
	}	
}


template<int SN>
__global__ void CompositeBasisRotKernel2(
	int64_t n_elements, int p_offset, int p_n, int p_total, bool enable_rot,
	const float* __restrict__ params,
	const float* __restrict__ faceNR,
	const float* __restrict__ rot_mean, // {P}x4
	const float* __restrict__ rot_basis, // {P}x4xK
	float* __restrict__ rot_tar // {P}x4
) {
	__shared__ float tile[SN];
	if (threadIdx.x < p_n) {
		tile[threadIdx.x] = params[p_offset + threadIdx.x];
	}
	__syncthreads();
	int64_t i_elem = (int64_t)blockIdx.x * blockDim.x + (int64_t)threadIdx.x;
	if (i_elem < n_elements) {
		float v[4];
		v[0] = rot_mean[i_elem * 4 + 0];
		v[1] = rot_mean[i_elem * 4 + 1];
		v[2] = rot_mean[i_elem * 4 + 2];
		v[3] = rot_mean[i_elem * 4 + 3];
		// do normalize
		float length = length4(v);
		v[0] = v[0] / length;
		v[1] = v[1] / length;
		v[2] = v[2] / length;
		v[3] = v[3] / length;
		if (enable_rot) {
			float loc_faceNR[9];
			//for (int i = 0; i < 9; i++) {
			//	loc_faceNR[i] = faceNR[9 * i_elem + i];
			//}
			
			// Eigen Matrix is col major !!
			loc_faceNR[0] = faceNR[i_elem * 9 + 0]; loc_faceNR[1] = faceNR[i_elem * 9 + 3]; loc_faceNR[2] = faceNR[i_elem * 9 + 6];
			loc_faceNR[3] = faceNR[i_elem * 9 + 1]; loc_faceNR[4] = faceNR[i_elem * 9 + 4]; loc_faceNR[5] = faceNR[i_elem * 9 + 7];
			loc_faceNR[6] = faceNR[i_elem * 9 + 2]; loc_faceNR[7] = faceNR[i_elem * 9 + 5]; loc_faceNR[8] = faceNR[i_elem * 9 + 8];
			
			float q_faceNR[4];
			float tmpv[4];
			matrix_to_quaternion(q_faceNR, loc_faceNR);
			tmpv[0] = v[0]; tmpv[1] = v[1]; tmpv[2] = v[2]; tmpv[3] = v[3];
			quaternion_raw_multiply(v, q_faceNR, tmpv);
		}
		//////////
		for (int ii = 0; ii < p_n; ii++) {
			float k = tile[ii];
			int real_i = ii + p_offset;
			v[0] += rot_basis[(i_elem * 4 + 0) * p_total + real_i] * k;
			v[1] += rot_basis[(i_elem * 4 + 1) * p_total + real_i] * k;
			v[2] += rot_basis[(i_elem * 4 + 2) * p_total + real_i] * k;
			v[3] += rot_basis[(i_elem * 4 + 3) * p_total + real_i] * k;
		}
		//// normalize quaternion
		length = length4(v);
		rot_tar[i_elem * 4 + 0] = v[0] / length;
		rot_tar[i_elem * 4 + 1] = v[1] / length;
		rot_tar[i_elem * 4 + 2] = v[2] / length;
		rot_tar[i_elem * 4 + 3] = v[3] / length;
	}
}




template<int SN>
__global__ void CompositeBasisRotKernel(
	int64_t n_elements, int p_offset, int p_n, int p_total,
	const float* __restrict__ params,
	const float* __restrict__ rot_mean, // {P}x4
	const float* __restrict__ rot_basis, // {P}x4xK
	float* __restrict__ rot_tar  // {P}x4
) {
	__shared__ float tile[SN];
	if (threadIdx.x < p_n) {
		tile[threadIdx.x] = params[p_offset + threadIdx.x];
	}
	__syncthreads();
	int64_t i_elem = (int64_t)blockIdx.x * blockDim.x + (int64_t)threadIdx.x;
	if (i_elem < n_elements) {
		float v[4];
		v[0] = rot_mean[i_elem * 4 + 0];
		v[1] = rot_mean[i_elem * 4 + 1];
		v[2] = rot_mean[i_elem * 4 + 2];
		v[3] = rot_mean[i_elem * 4 + 3];
		for (int ii = 0; ii < p_n; ii++) {
			float k = tile[ii];
			int real_i = ii + p_offset;
			v[0] += rot_basis[(i_elem * 4 + 0) * p_total + real_i] * k;
			v[1] += rot_basis[(i_elem * 4 + 1) * p_total + real_i] * k;
			v[2] += rot_basis[(i_elem * 4 + 2) * p_total + real_i] * k;
			v[3] += rot_basis[(i_elem * 4 + 3) * p_total + real_i] * k;
		}
		//// normalize quaternion
		float length = length4(v);
		rot_tar[i_elem * 4 + 0] = v[0] / length;
		rot_tar[i_elem * 4 + 1] = v[1] / length;
		rot_tar[i_elem * 4 + 2] = v[2] / length;
		rot_tar[i_elem * 4 + 3] = v[3] / length;
	}	
}

template<int P, int SN>
__global__ void CompositeBasisOpacityKernel(
	int64_t n_elements, int p_offset, int p_n, int p_total,
	const float* __restrict__ params,
	const float* __restrict__ opacity_mean, // {Px1}
	const float* __restrict__ opacity_basis, // {Px1}xK
	float* __restrict__ opacity_tar  // {Px1}
) {
	__shared__ float tile[SN];
	if (threadIdx.x < p_n) {
		tile[threadIdx.x] = params[p_offset + threadIdx.x];
	}
	__syncthreads();
	for (int i_loop = 0; i_loop < P; i_loop++) {
		int64_t i_elem = ((int64_t)blockIdx.x * P + i_loop) * blockDim.x + (int64_t)threadIdx.x;
		if (i_elem < n_elements) {
			float v = opacity_mean[i_elem];
			for (int ii = 0; ii < p_n; ii++) {
				float k = tile[ii];
				int real_i = ii + p_offset;
				v += opacity_basis[i_elem * p_total + real_i] * k;
			}
			v = sigmoid(v);
			opacity_tar[i_elem] = v;
			//opacity_tar[i_elem] = 0.5f;
		}
	}
}

///////////////////////////

template<int P, int SN>
__global__ void BlendTexKernel(
	int height, int width, int p_offset, int p_n, int p_total,
	float * __restrict__ basis, // HxWx3xk
	float * __restrict__ means, // HxWx3
	float * __restrict__ params,
	float scale, bool bgr, hipSurfaceObject_t surfobj
) {
	__shared__ float tile[SN];
	if (threadIdx.x < p_n) {
		tile[threadIdx.x] = params[p_offset + threadIdx.x];
	}
	__syncthreads();

	for (int i_loop = 0; i_loop < P; i_loop++) {
		int64_t id = ((int64_t)blockIdx.x * P + i_loop) * blockDim.x + (int64_t)threadIdx.x;
		int i = (int)(id % width);
		int j = (int)(id / width);

		if (j < height) {
			float4 ans;
			ans.x = ans.y = ans.z = 0;
			if (means) {
				if (bgr) {
					ans.z = means[id * 3 + 0] * scale;
					ans.y = means[id * 3 + 1] * scale;
					ans.x = means[id * 3 + 2] * scale;
				}
				else {
					ans.x = means[id * 3 + 0] * scale;
					ans.y = means[id * 3 + 1] * scale;
					ans.z = means[id * 3 + 2] * scale;
				}
			}
			for (int ii = 0; ii < p_n; ii++) {
				float k = tile[ii];
				int real_i = ii + p_offset;
				if (bgr) {
					ans.z += k * basis[(id * 3 + 0) * p_total + real_i] * scale;
					ans.y += k * basis[(id * 3 + 1) * p_total + real_i] * scale;
					ans.x += k * basis[(id * 3 + 2) * p_total + real_i] * scale;
				}
				else {
					ans.x += k * basis[(id * 3 + 0) * p_total + real_i] * scale;
					ans.y += k * basis[(id * 3 + 1) * p_total + real_i] * scale;
					ans.z += k * basis[(id * 3 + 2) * p_total + real_i] * scale;
				}
			}
			//ans.x = tile[0];
			//ans.y = tile[1];
			//ans.z = tile[2];
			ans.w = 1;
			//ans.x = 1; ans.y = 0.5; ans.z = 0.5; ans.w = 1;

			surf2Dwrite(ans, surfobj, (int)sizeof(float4) * i, j, MY_SURF_BOUNDARY_MODE);
		}
	}
}

__global__ void fillColorKernel(
	float* __restrict__ gpu_buffer,
	int n_points, float R, float G, float B
) {
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_id < n_points * 3) {
		float v = (thread_id % 3 == 0) ? R : ((thread_id % 3 == 1) ? G : B);
		gpu_buffer[thread_id] = v;
	}
}

__global__ void colorDepthKernel(
	float* __restrict__ canvas,
	float k, float b, bool inverse,
	int n_pixels
) {
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	float color[3];

	if (thread_id < n_pixels) {
		float v = canvas[0 * n_pixels + thread_id];
		float sv = v * k + b;
		if (inverse)
			sv = 1.f - sv;
		colormap_jet(sv, color);
		if (v == 0.f) { // set back to black
			color[0] = 0.f;
			color[1] = 0.f;
			color[2] = 0.f;
		}
		canvas[0 * n_pixels + thread_id] = color[0];
		canvas[1 * n_pixels + thread_id] = color[1];
		canvas[2 * n_pixels + thread_id] = color[2];
	}
}

template<int SN>
__global__ void computeDepthKernel(
	float* __restrict__ gpu_buffer,
	const float* __restrict__ pos,
	const float* __restrict__ modelview,
	int n_points
) {
	__shared__ float tile[SN * 3];
	int thread_id = blockIdx.x * SN + threadIdx.x;
#pragma unroll
	for (int i = 0; i < 3; i++) {
		int loc_id = (blockIdx.x * 3 + i) * SN + threadIdx.x;
		if (loc_id < n_points * 3) {
			tile[SN * i + threadIdx.x] = pos[loc_id];
		}
	}
	__syncthreads();

	float outpos[3];
	if (thread_id < n_points) {
		float posx, posy, posz;
		posx = tile[threadIdx.x * 3 + 0];
		posy = tile[threadIdx.x * 3 + 1];
		posz = tile[threadIdx.x * 3 + 2];		
		//outpos[0] = modelview[0] * posx + modelview[4] * posy + modelview[8] * posz + modelview[12];
		//outpos[1] = modelview[1] * posx + modelview[5] * posy + modelview[9] * posz + modelview[13];
		outpos[2] = modelview[2] * posx + modelview[6] * posy + modelview[10] * posz + modelview[14];
		//gpu_buffer[thread_id] = outpos[2]; // +z
	}

	__syncthreads(); // make sure the first task is done

	if (thread_id < n_points) {
		tile[threadIdx.x * 3 + 0] = outpos[2];
		tile[threadIdx.x * 3 + 1] = outpos[2];
		tile[threadIdx.x * 3 + 2] = outpos[2];
	}

	__syncthreads();

#pragma unroll
	for (int i = 0; i < 3; i++) {
		int loc_id = (blockIdx.x * 3 + i) * SN + threadIdx.x;
		if (loc_id < n_points * 3) {
			gpu_buffer[loc_id] = tile[SN * i + threadIdx.x];
		}
	}
}

template<int SN>
__global__ void visAnisotropyKernel(
	float* __restrict__ fill_color,
	const float* __restrict__ scale,
	int n_points, int max_rate
) {
	__shared__ float tile[SN * 3];
	int thread_id = blockIdx.x * SN + threadIdx.x;
#pragma unroll
	for (int i = 0; i < 3; i++) {
		int loc_id = (blockIdx.x * 3 + i) * SN + threadIdx.x;
		if (loc_id < n_points * 3) {
			tile[SN * i + threadIdx.x] = scale[loc_id];
		}
	}
	__syncthreads();

	float color = 0.f;

	if (thread_id < n_points) {
		float sx, sy, sz;
		sx = tile[threadIdx.x * 3 + 0];
		sy = tile[threadIdx.x * 3 + 1];
		sz = tile[threadIdx.x * 3 + 2];
		float min_s = sx;
		float max_s = sx;
		if (sy < min_s) min_s = sy;
		if (sy > max_s) max_s = sy;
		if (sz < min_s) min_s = sz;
		if (sz > max_s) max_s = sz;
		float rate = (max_s + 1e-12) / (min_s + 1e-12);
		color = (rate - 1.f) / (max_rate - 1.f);
	}

	__syncthreads(); // make sure the first task is done

	if (thread_id < n_points) {
		tile[threadIdx.x * 3 + 0] = color;
		tile[threadIdx.x * 3 + 1] = color;
		tile[threadIdx.x * 3 + 2] = color;
	}

	__syncthreads();

#pragma unroll
	for (int i = 0; i < 3; i++) {
		int loc_id = (blockIdx.x * 3 + i) * SN + threadIdx.x;
		if (loc_id < n_points * 3) {
			fill_color[loc_id] = tile[SN * i + threadIdx.x];
		}
	}

}
	

namespace cuGaussianSplatting {
	
	extern struct hipGraphicsResource * resources[];

	inline int nBlock(int n, int blockSize) {
		return (n + blockSize - 1) / blockSize;
	}

	inline int64_t nBlock(int64_t n, int64_t blockSize) {
		return (n + blockSize - 1) / blockSize;
	}

	void BlendTex(
		int height, int width, int p_offset, int p_n, int p_total,
		void* basis, void* means, void* params, int register_idx, 
		float scale, bool bgr
	) {
		hipError_t cudaStatus = hipSuccess;
		hipArray* arr_ptr1;

		cudaStatus = hipGraphicsMapResources(1, &resources[register_idx], 0);// 96 us
		if (cudaStatus != hipSuccess) throw std::runtime_error("hipGraphicsMapResources fail!\n");

		cudaStatus = hipGraphicsSubResourceGetMappedArray(&arr_ptr1, resources[register_idx], 0, 0);
		if (cudaStatus != hipSuccess) throw std::runtime_error("hipGraphicsSubResourceGetMappedArray fail!\n");

		{
			hipSurfaceObject_t surfObj = 0;
			struct hipResourceDesc resDesc;
			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = arr_ptr1;
			cudaStatus = hipCreateSurfaceObject(&surfObj, &resDesc);
			if (cudaStatus != hipSuccess) throw std::runtime_error("hipCreateSurfaceObject fail!\n");
			const unsigned int loop = 4;
			const unsigned int block = 512;
			unsigned int grid = nBlock(width * height, block * loop);

			if (p_n <= 256) {
				// input.stride, input.point_offset, color_offset,
				BlendTexKernel<loop,256> << <grid, block >> > (
					height,width,p_offset, p_n, p_total,
					(float*)basis,(float*)means,(float*)params,
					scale, bgr, surfObj
				);
			}
			else {
				throw std::runtime_error("no proper kernel!");
			}
			//cudaStatus = hipDeviceSynchronize();
			//if (cudaStatus != hipSuccess) 	throw std::runtime_error("debug fail12!\n");

			//cudaStatus = hipGetLastError();
			//if (cudaStatus != hipSuccess) throw std::runtime_error("BlendTexKernel fail!\n");
			
			cudaStatus = hipDestroySurfaceObject(surfObj);
			if (cudaStatus != hipSuccess) throw std::runtime_error("hipDestroySurfaceObject fail!\n");
		}

		cudaStatus = hipGraphicsUnmapResources(1, &resources[register_idx], 0);
		if (cudaStatus != hipSuccess) throw std::runtime_error("hipGraphicsUnmapResources fail!\n");
	}

	void CheckCudaError(int i);

	void fillColorCore(float* gpu_buffer, int n_points, const float* cpu_color3) {
		hipError_t cudaStatus = hipSuccess;		
		const unsigned int block = 512;
		unsigned int grid = nBlock(n_points * 3,block);
		fillColorKernel << <grid, block >> > (gpu_buffer, n_points, cpu_color3[0], cpu_color3[1], cpu_color3[2]);		
		//cudaStatus = hipDeviceSynchronize();
		//if (cudaStatus != hipSuccess) 	throw std::runtime_error("debug fail13!\n");
	}	

	void fillColorWithDepthCore(
		float * gpu_buffer, const float * pos, int n_points, const float * modelview
	) {
		hipError_t cudaStatus = hipSuccess;
		const unsigned int block = 256;
		unsigned int grid = nBlock(n_points, block);
		computeDepthKernel<block> <<<grid, block >>> (gpu_buffer, pos, modelview, n_points);
		//cudaStatus = hipDeviceSynchronize();
		//if (cudaStatus != hipSuccess) 	throw std::runtime_error("debug fail13!\n");
		
		// Determine temporary device storage requirements
		//void* d_temp_storage = NULL;
		//size_t   temp_storage_bytes = 0;
		//hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, gpu_buffer, minmaxbuffer, n_points, min_op, 1e12);

		//// Allocate temporary storage
		//hipMalloc(&d_temp_storage, temp_storage_bytes);

		//// Run reduction (min)
		//hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, gpu_buffer, minmaxbuffer, n_points, min_op, 1e12);
		//// Run reduction (max)
		//hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, gpu_buffer, minmaxbuffer + 1, n_points, max_op, -1e12);
		//hipFree(d_temp_storage);		
	}

	void ColorDepthCore(
		float* canvas, int width, int height, float k, float b, bool inverse
	) {
		int n_pixels = width * height;
		hipError_t cudaStatus = hipSuccess;
		const unsigned int block = 256;
		unsigned int grid = nBlock(n_pixels, block);
		colorDepthKernel<<<grid,block>>>(canvas, k, b, inverse,n_pixels);
		//cudaStatus = hipDeviceSynchronize();
		//if (cudaStatus != hipSuccess) 	throw std::runtime_error("debug fail14!\n");
	}

	void visAnisotropyCore(float* fill_color, const float* scale, int n_points, float max_rate) {
		hipError_t cudaStatus = hipSuccess;
		const unsigned int block = 256;
		unsigned int grid = nBlock(n_points , block);
		visAnisotropyKernel<block> << <grid, block >> > (fill_color, scale, n_points, max_rate);
		//cudaStatus = hipDeviceSynchronize();
		//if (cudaStatus != hipSuccess) 	throw std::runtime_error("debug fail14!\n");
	}	

	void CompositeBasisCore2(
		const float* params,
		const float* transfer,
		// 
		const float* shs_mean,
		const float* opacity_mean,
		const float* rot_mean,
		const float* scale_mean,
		//
		const float* shs_basis,
		const float* opacity_basis,
		const float* rot_basis,
		const float* scale_basis,
		//
		float* shs_tar,
		float* opacity_tar,
		float* rot_tar,
		float* scale_tar,
		int n_points, int n_total_dim, int n_compute_dim,
		bool enable_rot_geom , bool enable_rot_sh 
	) {
		hipError_t cudaStatus = hipSuccess;
		const unsigned int loop = 4;
		const unsigned int block = 512;
		unsigned int grid;
		if (n_compute_dim <= 256) {
			//CheckCudaError(0);
			
			// NOTICE, here in out is the same may have problem
			if (enable_rot_sh) {
				grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)32);
				CompositeBasisRotSHKernel << <grid, 32 * 3 >> > (
					n_points,transfer, shs_mean, shs_tar
				);
				CheckCudaError(0);
				grid = (unsigned int)nBlock((int64_t)n_points * 4 * 4 * 3, (int64_t)block * loop);
				CompositeBasisSHKernel_Inplace<loop, 256> << <grid, block >> > (
					n_points * 4 * 4 * 3, 0, n_compute_dim, n_total_dim,
					params, shs_tar, shs_basis
				);
				CheckCudaError(1);
			}
			else {
				grid = (unsigned int)nBlock((int64_t)n_points * 4 * 4 * 3, (int64_t)block * loop);
				CompositeBasisSHKernel<loop, 256> << <grid, block >> > (
					n_points * 4 * 4 * 3, 0, n_compute_dim, n_total_dim,
					params, shs_mean, shs_basis, shs_tar
				);
			}
			//CheckCudaError(1);
			grid = (unsigned int)nBlock((int64_t)n_points * 3, (int64_t)block * loop);
			CompositeBasisScaleKernel<loop, 256> << <grid, block >> > (
				n_points * 3, 0, n_compute_dim, n_total_dim,
				params, scale_mean, scale_basis, scale_tar
			);
			//CheckCudaError(2);
			grid = (unsigned int)nBlock((int64_t)n_points * 1, (int64_t)block * loop);
			CompositeBasisOpacityKernel<loop, 256> << <grid, block >> > (
				n_points * 1, 0, n_compute_dim, n_total_dim,
				params, opacity_mean, opacity_basis, opacity_tar
				);
			//CheckCudaError(3);
			grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)block);
			CompositeBasisRotKernel2<256> << <grid, block >> > (
				n_points, 0, n_compute_dim, n_total_dim, enable_rot_geom,
				params, transfer, rot_mean, rot_basis, rot_tar
			);
			//CheckCudaError(4);
		}
		else {
			throw std::runtime_error("no proper kernel!");
		}
		//cudaStatus = hipDeviceSynchronize();
		//if (cudaStatus != hipSuccess) 	throw std::runtime_error("debug fail12!\n");

		//cudaStatus = hipGetLastError();
		//if (cudaStatus != hipSuccess) throw std::runtime_error("CompositeBasisKernel fail!\n");
	}

	void CompositeBasisCore(
		const float* params,
		//
		const float* shs_mean,
		const float* opacity_mean,
		const float* rot_mean,
		const float* scale_mean,
		//
		const float* shs_basis,
		const float* opacity_basis,
		const float* rot_basis,
		const float* scale_basis,
		//
		float* shs_tar,
		float* opacity_tar,
		float* rot_tar,
		float* scale_tar,
		int n_points, int n_total_dim, int n_compute_dim
	) {
		hipError_t cudaStatus = hipSuccess;
		const unsigned int loop = 4;
		const unsigned int block = 512;
		unsigned int grid;
		if (n_compute_dim <= 256) {
			//CheckCudaError(0);
			grid = (unsigned int)nBlock((int64_t)n_points * 4 * 4 * 3, (int64_t)block * loop);
			CompositeBasisSHKernel<loop, 256> << <grid, block >> > (
				n_points * 4 * 4 * 3, 0, n_compute_dim, n_total_dim,
				params, shs_mean, shs_basis, shs_tar
				);
			//CheckCudaError(1);
			grid = (unsigned int)nBlock((int64_t)n_points * 3, (int64_t)block * loop);
			CompositeBasisScaleKernel<loop, 256> << <grid, block >> > (
				n_points * 3, 0, n_compute_dim, n_total_dim,
				params, scale_mean, scale_basis, scale_tar
				);
			//CheckCudaError(2);
			grid = (unsigned int)nBlock((int64_t)n_points * 1, (int64_t)block * loop);
			CompositeBasisOpacityKernel<loop, 256> << <grid, block >> > (
				n_points * 1, 0, n_compute_dim, n_total_dim,
				params, opacity_mean, opacity_basis, opacity_tar
				);
			//CheckCudaError(3);
			grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)block);
			CompositeBasisRotKernel<256> << <grid, block >> > (
				n_points, 0, n_compute_dim, n_total_dim,
				params, rot_mean, rot_basis, rot_tar
				);
			//CheckCudaError(4);
		}
		else {
			throw std::runtime_error("no proper kernel!");
		}		
		//cudaStatus = hipDeviceSynchronize();
		//if (cudaStatus != hipSuccess) 	throw std::runtime_error("debug fail12!\n");

		//cudaStatus = hipGetLastError();
		//if (cudaStatus != hipSuccess) throw std::runtime_error("CompositeBasisKernel fail!\n");
	}

	//

	void ScaleActivationCore(
		const float* src, float* tar, int n_elements, int tar_offset
	) {
		const unsigned int loop = 4;
		const unsigned int block = 512;
		unsigned int grid;
		grid = (unsigned int)nBlock((int64_t)n_elements * 3, (int64_t)block * loop);
		CopyScaleKernel<loop><<<grid, block >>>(
			n_elements * 3, src, tar + tar_offset * 3, true 
		);
	}
	void OpacityActivationCore(
		const float* src, float* tar, int n_elements, int tar_offset) {
		const unsigned int loop = 4;
		const unsigned int block = 512;
		unsigned int grid;
		grid = (unsigned int)nBlock((int64_t)n_elements * 1, (int64_t)block * loop);
		CopyOpacityKernel<loop><<<grid, block >>>(
			n_elements * 1, src, tar + tar_offset * 1, true
		);
	}
	void RotActivationCore(
		const float* src, float* tar, int n_elements, int tar_offset) {
		const unsigned int block = 256;
		unsigned int grid;
		grid = (unsigned int)nBlock((int64_t)n_elements, (int64_t)block);
		CopyRotKernel<block><<<grid, block >>>(
			n_elements, src, tar + tar_offset * 4, true
		);
	}

	void TransferPosCore(const float* mat4x3, const float* src, float* tar, int n_elements, int tar_offset) {
		const unsigned int block = 256;
		unsigned int grid;
		grid = (unsigned int)nBlock((int64_t)n_elements, (int64_t)block);
		TransferPosKernel<<< grid, block >>>(
			n_elements, mat4x3, src, tar + tar_offset * 3
		);
	}
	void TransferSHCore(const float* mat3x3, const float* src, float* tar, int n_elements, int tar_offset) {
		const unsigned int block = 96;
		unsigned int grid;
		grid = (unsigned int)nBlock((int64_t)n_elements, (int64_t)32);
		TransferSHKernel<<<grid, block >>>(
			n_elements, mat3x3, src, tar + tar_offset * 16 * 3
		);
	}
	void TransferRotActivationCore(const float* qrot4, const float* src, float* tar, int n_elements, int tar_offset) {
		const unsigned int block = 256;
		unsigned int grid;
		grid = (unsigned int)nBlock((int64_t)n_elements, (int64_t)block);
		TransferRotActivationKernel << <grid, block >> > (
			n_elements, qrot4, src, tar + tar_offset * 4
		);
	}

	void CompositeFUPipev1Core(
		const float* params, // 51(expr)
		//
		const float* pos_mean,
		const float* rot_mean,
		const float* scale_mean,
		const float* opacity_mean,
		const float* shs_mean,
		//
		const float* xyz_basis,
		const float* rot_basis,
		const float* scale_basis,
		const float* opacity_basis,
		const float* shs_basis,
		//
		float* pos_tar,
		float* shs_tar,
		float* opacity_tar,
		float* rot_tar,
		float* scale_tar,
		//
		int n_points, int n_total_dim, int n_compute_dim,
		int tar_offset
	) {
		hipError_t cudaStatus = hipSuccess;
		const unsigned int loop = 4;
		const unsigned int block = 512;
		//const unsigned int block2 = 128;
		unsigned int grid;

		if (n_compute_dim <= 256) {
			
			//// these params(opacity, scale) are not affected by complex mechanism ...
			grid = (unsigned int)nBlock((int64_t)n_points * 3, (int64_t)block * loop);
			CompositeBasisScaleKernel<loop, 256> << <grid, block >> > (
				n_points * 3, 0, n_compute_dim, n_total_dim,
				params, scale_mean, scale_basis, scale_tar + 3 * tar_offset
				);

			grid = (unsigned int)nBlock((int64_t)n_points * 1, (int64_t)block * loop);
			CompositeBasisOpacityKernel<loop, 256> << <grid, block >> > (
				n_points * 1, 0, n_compute_dim, n_total_dim,
				params, opacity_mean, opacity_basis, opacity_tar + 1 * tar_offset
			);

			//// pos

			grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)block);
			CompositeBasisPosKernel<256><<<grid,block>>>(
				n_points, 0, n_compute_dim, n_total_dim,
				params, pos_mean, xyz_basis, pos_tar + 3 * tar_offset
			);

			//// rot

			grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)block);
			CompositeBasisRotKernel<256> <<<grid, block >>>(
				n_points, 0, n_compute_dim, n_total_dim,
				params, rot_mean, rot_basis, rot_tar + 3 * tar_offset
			);

			//// sh
			if (n_compute_dim == 51 && n_total_dim == 51) {
				grid = (unsigned int)nBlock((int64_t)n_points * 4 * 4 * 3, (int64_t)64);
				CompositeBasisSHKernelAccN64T64<51> <<<grid, 64 >>>(
					(int64_t)n_points * 4 * 4 * 3, params, shs_mean, shs_basis, shs_tar + tar_offset * 16 * 3 
				);
			}
			else {
				grid = (unsigned int)nBlock((int64_t)n_points * 4 * 4 * 3, (int64_t)block * loop);
				CompositeBasisSHKernel<loop, 256> <<<grid, block >>> (
					(int64_t)n_points * 4 * 4 * 3, 0, n_compute_dim, n_total_dim,
					params, shs_mean, shs_basis, shs_tar + tar_offset * 16 * 3
				);
			}
		}
		else {
			throw std::runtime_error("no proper kernel!");
		}

	}

	void CompositeNewPipev1Core(
		const float* params, // 100(expr) + 2(eyelid) + 36(pos)
		const float* node_transfer, // 5*4*4
		//
		const float* pos_mean,
		const float* rot_mean,
		const float* scale_mean,
		const float* opacity_mean,
		const float* shs_mean,
		//
		const float* xyz_basis,
		const float* rot_basis,
		const float* scale_basis,
		const float* opacity_basis,
		const float* shs_basis,
		//
		const float* rot_d,
		const float* pos_t,
		const float* W,
		const float* eyelid,
		//
		float* pos_tar,
		float* shs_tar,
		float* opacity_tar,
		float* rot_tar,
		float* scale_tar,
		//
		float ** buffer, size_t * buffer_size,
		bool enable_deform_rot, bool enable_deform_rot_sh,
		bool enable_trans_rot, bool enable_trans_rot_sh,
		int n_points, int n_total_dim, int n_compute_dim,
		int tar_offset, bool force_50
	) {
		hipError_t cudaStatus = hipSuccess;
		const unsigned int loop = 4;
		const unsigned int block = 512;
		const unsigned int block2 = 128;
		unsigned int grid;
		if (n_compute_dim + 36 <= 256) {
			
			//CheckCudaError(332);
			//// these params(opacity, scale) are not affected by complex mechanism ...
			grid = (unsigned int)nBlock((int64_t)n_points * 3, (int64_t)block * loop);
			CompositeBasisScaleKernel<loop, 256> << <grid, block >> > (
				n_points * 3, 0, n_compute_dim, n_total_dim,
				params, scale_mean, scale_basis, scale_tar + 3 * tar_offset
			);
			
			grid = (unsigned int)nBlock((int64_t)n_points * 1, (int64_t)block * loop);
			CompositeBasisOpacityKernel<loop, 256> << <grid, block >> > (
				n_points * 1, 0, n_compute_dim, n_total_dim,
				params, opacity_mean, opacity_basis, opacity_tar + 1 * tar_offset
			);
			//CheckCudaError(333);
			//// 1. pos (expr + pose)
			grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)block);
			if (force_50) {
				CompositeBasisPosKernel<256> << <grid, block >> > (
					n_points, 0, n_compute_dim, n_total_dim,
					100 + 2, 36,
					params, pos_mean, xyz_basis, pos_t, pos_tar + 3 * tar_offset
				);
			}
			else {
				CompositeBasisPosKernel<256> << <grid, block >> > (
					n_points, 0, 100, 100,
					100 + 2, 36,
					params, pos_mean, xyz_basis, pos_t, pos_tar + 3 * tar_offset
				);
			}

			// allocate buffer
			int n_elem1 = nBlock(12 * n_points,32) * 32; // for byte alignment
			int n_elem2 = nBlock(4 * n_points,32) * 32;
			int n_elem3 = nBlock(4 * n_points,32) * 32;
			float* ptr_root;
			float* ptr_TN;
			float* ptr_DR;
			if (buffer && buffer_size) {
				if (*buffer == nullptr) {
					// allocate first time
					hipMalloc(buffer, sizeof(float) * (n_elem1 + n_elem2 + n_elem3));
					*buffer_size = n_elem1 + n_elem2 + n_elem3;
				}
				else if ((n_elem1 + n_elem2 + n_elem3) != *buffer_size) {
					// re-allocate
					hipFree(*buffer);
					hipMalloc(buffer, sizeof(float) * (n_elem1 + n_elem2 + n_elem3));
					*buffer_size = n_elem1 + n_elem2 + n_elem3;
				}
				ptr_root = *buffer;
			}
			else {
				hipMalloc(&ptr_root, sizeof(float) * (n_elem1 + n_elem2 + n_elem3));
			}
			ptr_TN = ptr_root + n_elem1;
			ptr_DR = ptr_TN + n_elem2;

			// 2. skinning and compute transfer
			grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)block2);
			CompositeT<block2><<<grid,block2>>>(
				n_points, W, node_transfer, ptr_root, ptr_TN
			);

			bool compute_deform_rot = enable_deform_rot || enable_deform_rot_sh;
			if (compute_deform_rot && !rot_d) {
				static int warning_flag = 0; // warning only once !
				if (warning_flag == 0) {
					printf("warning! rot_d is not available\n");
					warning_flag = 1;
				}
			}			
			compute_deform_rot = rot_d && compute_deform_rot;

			// 3. [optional]
			if (compute_deform_rot) {
				grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)block);
				CompositeDeformRot<256><<<grid,block>>>(
					//n_points, 0, n_compute_dim, n_total_dim, true
					n_points, 0, 100, 100, true,
					params, rot_d, ptr_DR
				);
			}

			//CheckCudaError(334);

			// 4. apply transfer and add eyelid
			grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)block2);
			RotPosAndAddEyelidKernel_Inplace<block2><<<grid,block2>>>(
				n_points, 100,
				params, ptr_root, eyelid, pos_tar + 3 * tar_offset
			);

			//////////////////

			// 5. rot: apply deform rot and basis
			grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)block);
			CompositeBasisRotKernel3<256> << <grid, block >> > ( 
				n_points, 0, n_compute_dim, n_total_dim, compute_deform_rot && enable_deform_rot,
				params, rot_mean, rot_basis, ptr_DR, rot_tar + 4 * tar_offset
			);
			//CheckCudaError(335);

			// 6. rot: apply trans rot 
			if (enable_trans_rot) {
				CompositeRotKernel_Inplace<256> << <grid, block >> > (
					n_points, params, ptr_TN, rot_tar + 4 * tar_offset
				);
				//CheckCudaError(336);
			}

			//////////////////

			// 7. sh: apply deform rot 
			if (compute_deform_rot && enable_deform_rot_sh) {
				grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)32);
				TransferSHKernelQuat<< <grid, 96 >> > (
					n_points, ptr_DR, shs_mean, shs_tar + tar_offset * 16 * 3
				);
			}
			else {
				hipMemcpy(shs_tar + tar_offset * 16 * 3, shs_mean, sizeof(float) * n_points * 16 * 3, hipMemcpyDeviceToDevice);
			}
			//CheckCudaError(337);
			
			// 8. sh: add basis			
			
			if (n_compute_dim == 50 && n_total_dim == 50) {
				// acceleration but assert n_compute_dim == 50, n_total_dim == 50
				
				//grid = (unsigned int)nBlock((int64_t)n_points * 4 * 4 * 3, (int64_t)32);
				//CompositeBasisSHKernelAccN32T256_Inplace<50> << <grid, 256 >> > (
				//	(int64_t)n_points * 4 * 4 * 3, params, shs_tar + tar_offset * 16 * 3, shs_basis
				//);	

				//grid = (unsigned int)nBlock((int64_t)n_points * 4 * 4 * 3, (int64_t)32);
				//CompositeBasisSHKernelAccN32T64_Inplace<50> << <grid, 64 >> > (
				//	(int64_t)n_points * 4 * 4 * 3, params, shs_tar + tar_offset * 16 * 3, shs_basis
				//);

				grid = (unsigned int)nBlock((int64_t)n_points * 4 * 4 * 3, (int64_t)64);
				CompositeBasisSHKernelAccN64T64_Inplace<50> << <grid, 64 >> > (
					(int64_t)n_points * 4 * 4 * 3, params, shs_tar + tar_offset * 16 * 3, shs_basis
				);

				//grid = (unsigned int)nBlock((int64_t)n_points * 4 * 4 * 3, (int64_t)128);
				//CompositeBasisSHKernelAccN128T128_Inplace<50> << <grid, 128 >> > (
				//	(int64_t)n_points * 4 * 4 * 3, params, shs_tar + tar_offset * 16 * 3, shs_basis
				//);				

			}
			else {
				grid = (unsigned int)nBlock((int64_t)n_points * 4 * 4 * 3, (int64_t)block * loop);
				CompositeBasisSHKernel_Inplace<loop, 256> << <grid, block >> > (
					(int64_t)n_points * 4 * 4 * 3, 0, n_compute_dim, n_total_dim,
					params, shs_tar + tar_offset * 16 * 3, shs_basis
				);
			}			

			//CheckCudaError(338);
			// 9. sh: apply trans rot 
			if (enable_trans_rot_sh) {
				grid = (unsigned int)nBlock((int64_t)n_points, (int64_t)32);
				TransferSHKernelQuat_Inplace << <grid, 96 >> > (
					n_points, ptr_TN, shs_tar + tar_offset * 16 * 3
				);
			}
			//CheckCudaError(339);

			if (buffer && buffer_size) { /* nothing to do, no need to allocate and release each time */ }
			else { hipFree(ptr_root); }
		}
		else {
			throw std::runtime_error("no proper kernel!");
		}
	}

}
